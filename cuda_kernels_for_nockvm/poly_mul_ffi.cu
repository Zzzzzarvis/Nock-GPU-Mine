// poly_mul_ffi.cu - FFI implementation for polynomial multiplication CUDA module

#include "poly_mul_ffi.h"
#include "poly_mul.cu" // Include the actual CUDA kernel implementation
#include <hip/hip_runtime.h>
#include <cstdio> // For printf in case of errors

// Helper to check CUDA errors and map to FFI error codes
static CudaFFIErrorCode check_cuda_error(hipError_t err, const char* operation) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error during %s: %s\n", operation, hipGetErrorString(err));
        // Map specific CUDA errors to FFI errors if needed, otherwise a general one
        if (err == hipErrorOutOfMemory) return CUDA_ERROR_FFI_ALLOC;
        if (err == hipErrorInvalidValue) return CUDA_ERROR_FFI_INVALID_ARGS;
        // Add more mappings as necessary
        return CUDA_ERROR_FFI_KERNEL_LAUNCH; // General kernel/runtime error
    }
    return CUDA_SUCCESS_FFI;
}

extern "C" CudaFFIErrorCode cuda_poly_mul(const uint64_t* h_poly_a, int len_a,
                                          const uint64_t* h_poly_b, int len_b,
                                          uint64_t* h_poly_res, int len_res) {
    if (!h_poly_a || !h_poly_b || !h_poly_res || len_a <= 0 || len_b <= 0) {
        return CUDA_ERROR_FFI_INVALID_ARGS;
    }
    if (len_res != len_a + len_b - 1 && !(len_a == 1 && len_b == 1 && len_res ==1)) { // handle 1x1 case for testing
         if (len_res != len_a + len_b -1) {
            fprintf(stderr, "Error: Result polynomial length is incorrect. Expected %d, got %d\n", len_a + len_b - 1, len_res);
            return CUDA_ERROR_FFI_INVALID_ARGS;
        }
    }

    uint64_t* d_poly_a = nullptr;
    uint64_t* d_poly_b = nullptr;
    uint64_t* d_poly_res = nullptr;
    CudaFFIErrorCode ffi_err_code = CUDA_SUCCESS_FFI;
    hipError_t cuda_err;

    // 1. Allocate memory on the device
    cuda_err = hipMalloc((void**)&d_poly_a, len_a * sizeof(uint64_t));
    if ((ffi_err_code = check_cuda_error(cuda_err, "hipMalloc d_poly_a")) != CUDA_SUCCESS_FFI) goto cleanup;
    cuda_err = hipMalloc((void**)&d_poly_b, len_b * sizeof(uint64_t));
    if ((ffi_err_code = check_cuda_error(cuda_err, "hipMalloc d_poly_b")) != CUDA_SUCCESS_FFI) goto cleanup;
    cuda_err = hipMalloc((void**)&d_poly_res, len_res * sizeof(uint64_t));
    if ((ffi_err_code = check_cuda_error(cuda_err, "hipMalloc d_poly_res")) != CUDA_SUCCESS_FFI) goto cleanup;

    // 2. Copy data from host to device
    cuda_err = hipMemcpy(d_poly_a, h_poly_a, len_a * sizeof(uint64_t), hipMemcpyHostToDevice);
    if ((ffi_err_code = check_cuda_error(cuda_err, "hipMemcpy h_poly_a to d_poly_a")) != CUDA_SUCCESS_FFI) goto cleanup;
    cuda_err = hipMemcpy(d_poly_b, h_poly_b, len_b * sizeof(uint64_t), hipMemcpyHostToDevice);
    if ((ffi_err_code = check_cuda_error(cuda_err, "hipMemcpy h_poly_b to d_poly_b")) != CUDA_SUCCESS_FFI) goto cleanup;

    // 3. Define kernel launch parameters
    // For poly_mul_kernel, each thread calculates one coefficient of the result.
    // The number of threads should be equal to len_res.
    int threads_per_block = 256;
    int blocks_per_grid = (len_res + threads_per_block - 1) / threads_per_block;

    // 4. Launch the kernel
    poly_mul_kernel<<<blocks_per_grid, threads_per_block>>>(d_poly_a, len_a, d_poly_b, len_b, d_poly_res, len_res);
    cuda_err = hipGetLastError(); // Check for errors during kernel launch
    if ((ffi_err_code = check_cuda_error(cuda_err, "poly_mul_kernel launch")) != CUDA_SUCCESS_FFI) goto cleanup;
    
    // Synchronize to ensure kernel completion before copying back (optional for hipMemcpyDeviceToHost if not on default stream and using events)
    cuda_err = hipDeviceSynchronize();
    if ((ffi_err_code = check_cuda_error(cuda_err, "hipDeviceSynchronize after kernel")) != CUDA_SUCCESS_FFI) goto cleanup;

    // 5. Copy data from device to host
    cuda_err = hipMemcpy(h_poly_res, d_poly_res, len_res * sizeof(uint64_t), hipMemcpyDeviceToHost);
    if ((ffi_err_code = check_cuda_error(cuda_err, "hipMemcpy d_poly_res to h_poly_res")) != CUDA_SUCCESS_FFI) goto cleanup;

cleanup:
    // 6. Free device memory
    if (d_poly_a) hipFree(d_poly_a);
    if (d_poly_b) hipFree(d_poly_b);
    if (d_poly_res) hipFree(d_poly_res);

    return ffi_err_code;
}

